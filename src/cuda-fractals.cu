#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <thrust/complex.h>
#include "fractals.h"

//These won't work/ need to be modifiied for the cuda version
//#include "grids.h"
//#include "fractals.h"

/*
 * Macro for checking CUDA errors
 */
#define CHECK(call) \
{ \
    const hipError_t error = call; \
    if(error != hipSuccess){ \
        fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__); \
        fprintf(stderr, "Code: %d, Reason: %s\n", error, hipGetErrorString(error)); \
    } \
}

#ifndef CBASE
#define CBASE double
#endif

#define BLOCK_SIZE_X 16
#define BLOCK_SIZE_Y 16

// this ain't pretty, but this logic should never change for kernels
#define SET_ROW_COL \
    const size_t row = blockIdx.y * blockDim.y + threadIdx.y; \
    const size_t col = blockIdx.x * blockDim.x + threadIdx.x; \
    if(row >= rows || col >= cols) return


__device__
thrust::complex<CBASE> grid_to_complex(const thrust::complex<CBASE> lower_left, const thrust::complex<CBASE> upper_right, const size_t row, const size_t col, const size_t rows, const size_t cols){
    const CBASE x_min = lower_left.real();
    const CBASE x_max = upper_right.real();
    const CBASE y_min = lower_left.imag();
    const CBASE y_max = upper_right.imag();

    const CBASE x_step = (x_max - x_min) / (CBASE)cols;
    const CBASE y_step = (y_max - y_min) / (CBASE)rows;

    const CBASE x = x_min + col * x_step;
    const CBASE y = y_min + row * y_step;
    const thrust::complex<CBASE> z(x,y);
    return z;
}

__device__
byte mandelbrot(const thrust::complex<CBASE> z0, const byte max_iterations){
    thrust::complex<CBASE> z = z0;
    byte iteration = 0;
    while(thrust::abs(z) <= 2 && iteration < max_iterations){
        z = z*z + z0;
        iteration++;
    }
    return iteration;
}

__global__
void mandelbrot_kernel(byte* grid_data, const byte max_iterations, const thrust::complex<CBASE> lower_left, const thrust::complex<CBASE> upper_right, const size_t rows, const size_t cols){
    SET_ROW_COL;

    const auto z = grid_to_complex(lower_left, upper_right, row, col, rows, cols);

    grid_data[row*cols + col] = mandelbrot(z, max_iterations);
}

__device__ 
byte tricorn(const thrust::complex<CBASE> z0, const byte max_iterations){
    thrust::complex<CBASE> z = z0;
    byte iteration = 0;
    while(thrust::abs(z) <= 2 && iteration < max_iterations){
        z = thrust::conj(z*z) + z0;
        iteration++;
    }
    return iteration;
}

__global__
void tricorn_kernel(byte* grid_data, const byte max_iterations, const thrust::complex<CBASE> lower_left, const thrust::complex<CBASE> upper_right, const size_t rows, const size_t cols){
    SET_ROW_COL;

    const auto z = grid_to_complex(lower_left, upper_right, row, col, rows, cols);

    grid_data[row*cols + col] = tricorn(z, max_iterations);
}

__device__
byte burning_ship(const thrust::complex<CBASE> z0, const byte max_iterations){
    thrust::complex<CBASE> z = z0;
    thrust::complex<CBASE> z_mod;
    byte iteration = 0;
    while(thrust::abs(z) <= 2 && iteration < max_iterations){
        z_mod = thrust::complex<CBASE>(fabs(z.real()), fabs(z.imag()));
        z = z_mod * z_mod + z0;
        iteration++;
    }
    return iteration;
}

__global__
void burning_ship_kernel(byte* grid_data, const byte max_iterations, const thrust::complex<CBASE> lower_left, const thrust::complex<CBASE> upper_right, const size_t rows, const size_t cols){
    SET_ROW_COL;

    const auto z = grid_to_complex(lower_left, upper_right, row, col, rows, cols);

    grid_data[row*cols + col] = burning_ship(z, max_iterations);
}

__device__
byte multibrot(const thrust::complex<CBASE> z0, const byte max_iterations, const double d){
    thrust::complex<CBASE> z = z0;
    byte iteration = 0;
    while(thrust::abs(z) <= 2 && iteration < max_iterations){
        z = thrust::pow(z, d) + z0;
        iteration++;
    }
    return iteration;
}

__global__
void multibrot_kernel(byte* grid_data, const double degree, const byte max_iterations, const thrust::complex<CBASE> lower_left, const thrust::complex<CBASE> upper_right, const size_t rows, const size_t cols){
    SET_ROW_COL;

    const auto z = grid_to_complex(lower_left, upper_right, row, col, rows, cols);

    grid_data[row*cols + col] = multibrot(z, max_iterations, degree);
}

__device__
byte multicorn(const thrust::complex<CBASE> z0, const byte max_iterations, const double d){
    thrust::complex<CBASE> z = z0;
    byte iteration = 0;
    while(thrust::abs(z) <= 2 && iteration < max_iterations){
        z = thrust::conj(thrust::pow(z, d)) + z0;
        iteration++;
    }
    return iteration;
}

__global__
void multicorn_kernel(byte* grid_data, const double degree, const byte max_iterations, const thrust::complex<CBASE> lower_left, const thrust::complex<CBASE> upper_right, const size_t rows, const size_t cols){
    SET_ROW_COL;

    const auto z = grid_to_complex(lower_left, upper_right, row, col, rows, cols);

    grid_data[row*cols + col] = multicorn(z, max_iterations, degree);
}

__device__
byte julia(const thrust::complex<CBASE> z0, const byte max_iterations, const thrust::complex<CBASE> c, const double R){
    thrust::complex<CBASE> z = z0;
    byte iteration = 0;
    while(thrust::abs(z) <= R && iteration < max_iterations){
        z = z*z + c;
        iteration++;
    }
    return iteration;
}

__global__
void julia_kernel(byte* grid_data, const thrust::complex<CBASE> constant, const double radius, const byte max_iterations, const thrust::complex<CBASE> lower_left, const thrust::complex<CBASE> upper_right, const size_t rows, const size_t cols){
    SET_ROW_COL;

    const auto z = grid_to_complex(lower_left, upper_right, row, col, rows, cols);

    grid_data[row*cols + col] = julia(z, max_iterations, constant, radius);
}

extern "C" {
void mandelbrot_grid(grid_t* grid, const grid_gen_params* params){
    const size_t size = grid->size;
    const size_t rows = grid->y;
    const size_t cols = grid->x;
    const byte max_iterations = grid->max_iterations;
    thrust::complex<CBASE> lower_left(grid->lower_left.re, grid->lower_left.im);
    thrust::complex<CBASE> upper_right(grid->upper_right.re, grid->upper_right.im);

    byte* d_grid_data;
    CHECK(hipMalloc(&d_grid_data, size*sizeof(byte)));
    //TODO: find good sizes
    dim3 block_size(BLOCK_SIZE_X, BLOCK_SIZE_Y);
    //dim3 grid_size(0,0);
    dim3 grid_size((cols + block_size.x - 1) / block_size.x, (rows + block_size.y - 1) / block_size.y);
    mandelbrot_kernel<<<grid_size, block_size>>>(d_grid_data, max_iterations, lower_left, upper_right, rows, cols);
    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(grid->data, d_grid_data, size*sizeof(byte), hipMemcpyDeviceToHost));

    CHECK(hipFree(d_grid_data));
    CHECK(hipDeviceReset());
}

void tricorn_grid(grid_t* grid, const grid_gen_params* params){
    const size_t size = grid->size;
    const size_t rows = grid->y;
    const size_t cols = grid->x;
    const byte max_iterations = grid->max_iterations;
    thrust::complex<CBASE> lower_left(grid->lower_left.re, grid->lower_left.im);
    thrust::complex<CBASE> upper_right(grid->upper_right.re, grid->upper_right.im);

    byte* d_grid_data;
    CHECK(hipMalloc(&d_grid_data, size*sizeof(byte)));
    dim3 block_size(BLOCK_SIZE_X, BLOCK_SIZE_Y);
    dim3 grid_size((cols + block_size.x - 1) / block_size.x, (rows + block_size.y - 1) / block_size.y);
    tricorn_kernel<<<grid_size, block_size>>>(d_grid_data, max_iterations, lower_left, upper_right, rows, cols);
    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(grid->data, d_grid_data, size*sizeof(byte), hipMemcpyDeviceToHost));

    CHECK(hipFree(d_grid_data));
    CHECK(hipDeviceReset());
}

void burning_ship_grid(grid_t* grid, const grid_gen_params* params){
    const size_t size = grid->size;
    const size_t rows = grid->y;
    const size_t cols = grid->x;
    const byte max_iterations = grid->max_iterations;
    thrust::complex<CBASE> lower_left(grid->lower_left.re, grid->lower_left.im);
    thrust::complex<CBASE> upper_right(grid->upper_right.re, grid->upper_right.im);

    byte* d_grid_data;
    CHECK(hipMalloc(&d_grid_data, size*sizeof(byte)));
    dim3 block_size(BLOCK_SIZE_X, BLOCK_SIZE_Y);
    dim3 grid_size((cols + block_size.x - 1) / block_size.x, (rows + block_size.y - 1) / block_size.y);
    burning_ship_kernel<<<grid_size, block_size>>>(d_grid_data, max_iterations, lower_left, upper_right, rows, cols);
    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(grid->data, d_grid_data, size*sizeof(byte), hipMemcpyDeviceToHost));

    CHECK(hipFree(d_grid_data));
    CHECK(hipDeviceReset());
}

void multibrot_grid(grid_t* grid, const grid_gen_params* params){
    const size_t size = grid->size;
    const size_t rows = grid->y;
    const size_t cols = grid->x;
    const byte max_iterations = grid->max_iterations;
    const double degree = params->degree;
    thrust::complex<CBASE> lower_left(grid->lower_left.re, grid->lower_left.im);
    thrust::complex<CBASE> upper_right(grid->upper_right.re, grid->upper_right.im);

    byte* d_grid_data;
    CHECK(hipMalloc(&d_grid_data, size*sizeof(byte)));
    dim3 block_size(BLOCK_SIZE_X, BLOCK_SIZE_Y);
    dim3 grid_size((cols + block_size.x - 1) / block_size.x, (rows + block_size.y - 1) / block_size.y);
    multibrot_kernel<<<grid_size, block_size>>>(d_grid_data, degree, max_iterations, lower_left, upper_right, rows, cols);
    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(grid->data, d_grid_data, size*sizeof(byte), hipMemcpyDeviceToHost));

    CHECK(hipFree(d_grid_data));
    CHECK(hipDeviceReset());
}

void multicorn_grid(grid_t* grid, const grid_gen_params* params){
    const size_t size = grid->size;
    const size_t rows = grid->y;
    const size_t cols = grid->x;
    const byte max_iterations = grid->max_iterations;
    const double degree = params->degree;
    thrust::complex<CBASE> lower_left(grid->lower_left.re, grid->lower_left.im);
    thrust::complex<CBASE> upper_right(grid->upper_right.re, grid->upper_right.im);

    byte* d_grid_data;
    CHECK(hipMalloc(&d_grid_data, size*sizeof(byte)));
    dim3 block_size(BLOCK_SIZE_X, BLOCK_SIZE_Y);
    dim3 grid_size((cols + block_size.x - 1) / block_size.x, (rows + block_size.y - 1) / block_size.y);
    multicorn_kernel<<<grid_size, block_size>>>(d_grid_data, degree, max_iterations, lower_left, upper_right, rows, cols);
    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(grid->data, d_grid_data, size*sizeof(byte), hipMemcpyDeviceToHost));

    CHECK(hipFree(d_grid_data));
    CHECK(hipDeviceReset());
}

void julia_grid(grid_t* grid, const grid_gen_params* params){
    thrust::complex<CBASE> constant(params->cr.constant.re, params->cr.constant.im);
    const double radius = params->cr.radius;
    const size_t size = grid->size;
    const size_t rows = grid->y;
    const size_t cols = grid->x;
    const byte max_iterations = grid->max_iterations;
    thrust::complex<CBASE> lower_left(grid->lower_left.re, grid->lower_left.im);
    thrust::complex<CBASE> upper_right(grid->upper_right.re, grid->upper_right.im);

    byte* d_grid_data;
    CHECK(hipMalloc(&d_grid_data, size*sizeof(byte)));
    dim3 block_size(BLOCK_SIZE_X, BLOCK_SIZE_Y);
    dim3 grid_size((cols + block_size.x - 1) / block_size.x, (rows + block_size.y - 1) / block_size.y);
    julia_kernel<<<grid_size, block_size>>>(d_grid_data, constant, radius, max_iterations, lower_left, upper_right, rows, cols);
    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(grid->data, d_grid_data, size*sizeof(byte), hipMemcpyDeviceToHost));

    CHECK(hipFree(d_grid_data));
    CHECK(hipDeviceReset());
}
}
